#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void matXvec(int* mat, int* vec, int* prod)
{
	//mat[threadIdx.x + blockDim.x * threadIdx.y] = mat[threadIdx.x + blockDim.x * threadIdx.y] * vec[threadIdx.x];
	//int ty = threadIdx.y;
	//int tx = threadIdx.x;
	//atomicAdd(&prod[ty], mat[ty * 32 + tx]);
	atomicAdd(&prod[threadIdx.y], vec[threadIdx.x] * mat[threadIdx.y * blockDim.x + threadIdx.x]);
}

int* read_array(const char* filename, int len) {
	int *x = (int*) malloc(len * sizeof(int));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%d", &x[i]);
	}
	fclose(fp);
	return x;
}

int main(int argc, char *argv[]) {
	if (argc != 1) {
		printf("Invalid argument Usage: ./problem1");
		return -1;
	}

	const int rowWidth=32;
	const int colWidth=16;	
	int *hA = read_array("inputA.inp",rowWidth*colWidth );
	int *hB = read_array("inputB.inp", rowWidth);
	int *hC = (int*) malloc(colWidth * sizeof(int));
	int *refC = (int*) malloc(colWidth * sizeof(int));
	// TODO - allocate host memory for refC (you have to figure out how much)
	// The skeleton currently segfaults because refC is accessed without allocation

	// TODO do a reference host implementation (Ch) here. ie populate answer in refC
	for(int i = 0; i < colWidth; i++)
	{
		refC[i] = 0;
		for(int j = 0; j < rowWidth; j++)
		{
			refC[i] = refC[i] + hA[j + rowWidth * i] * hB[j];
		}
		//printf("refC[%d]=%d\n",i, refC[i]);
	}

	int *dA, *dB, *dC;
	// TODO allocate device memory for dA,dB and dC
	hipMalloc((void**)&dA, sizeof(int) * rowWidth * colWidth);
	hipMalloc((void**)&dB, sizeof(int) * rowWidth);
	hipMalloc((void**)&dC, sizeof(int) * colWidth);

	// TODO copy data from host to GPU 
	hipMemcpy(dA, hA, sizeof(int) * rowWidth * colWidth, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(int) * rowWidth, hipMemcpyHostToDevice);

	// TODO call your kernel
	dim3 threadsPerBlock(rowWidth, colWidth);
	matXvec<<<1, threadsPerBlock>>>(dA, dB, dC);

	// TODO copyback results
	hipMemcpy(hC, dC, sizeof(int) * colWidth, hipMemcpyDeviceToHost);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	float Error=0;

	for(int i=0;i<colWidth;i++)
		Error+=(hC[i]-refC[i])*(hC[i]-refC[i]);
	printf("%f\n%d",sqrt(Error),hC[colWidth-1]);

	free(refC);
	free(hB);
	free(hA);

	return 0;
}
