#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addTidBid(int* data)
{
	data[threadIdx.x + blockIdx.x * blockDim.x] = (threadIdx.x + blockIdx.x);
	//printf("i am tid %d blockDim %d blockIdx %d\n", threadIdx.x, blockDim.x, blockIdx.x);
}

int main()
{
	const int numElems = 16;
	int *hArray = (int *) malloc(numElems * sizeof(int));
	int* dArray;													//initialize pointer on host
	hipMalloc((void **)&dArray, sizeof(int) * numElems);	//allocate memory on device
	hipMemset(dArray, 0, numElems * sizeof(int));					//initilize them to zero
	addTidBid<<<2,8>>>(dArray);										//kernel launched 2blocks 8threads
	hipMemcpy(hArray, dArray, sizeof(int) * numElems, hipMemcpyDeviceToHost);
	hipFree(dArray);
	for (int i = 0; i < numElems; i++)
		std::cout << hArray[i] << std::endl;

	free(hArray);
	return(0);
}
