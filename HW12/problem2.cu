#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>

void prefix_scan(float *in, float *out, int N)
{
	thrust::host_vector<float> H(N);
	for (int i = 0; i < N; i++)
	{
		H[i] = in[i];
	}
	thrust::device_vector<float> D = H;
	thrust::inclusive_scan(D.begin(), D.end(), D.begin());
	thrust::copy(D.begin(), D.end(), out);
	return;
}