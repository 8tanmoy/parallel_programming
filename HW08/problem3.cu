#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

__global__ void MatMult(double *dA, double *dB, double *dC, int nRows, int nInnerDimension, int nCols, int TileSize)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x; 
	int ty = threadIdx.y;
	int aBegin = nRows * TileSize * by;
	int aEnd   = aBegin + nRows - 1;
	int aStep  = TileSize;
	int bBegin = bx * TileSize;
	int bStep  = TileSize * nInnerDimension;
	double Csub = 0.0;

	volatile __shared__ double As[32][32];
	volatile __shared__ double Bs[32][32];
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b +=bStep)
	{
		As[ty][tx] = dA[a + nRows * ty + tx];
		Bs[ty][tx] = dB[b + nInnerDimension * ty + tx];

		__syncthreads();

		for (int k = 0; k < TileSize; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
		}

		__syncthreads();
		//__threadfence_block();
	}
	int c = nInnerDimension * TileSize * by + TileSize * bx;
	dC[c + nInnerDimension * ty + tx] = Csub;
}

double* read_array(const char* filename, int len) {
	double *x = (double*) malloc(len * sizeof(double));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%lf", &x[i]);
	}
	fclose(fp);
	return x;
}

void computeOnDevice(double* hA,double* hB, double* hC, int nRows,
	int nInnerDimension,int nCols, int tileSize, float* incTime );

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv) 
{
	if(argc!=5)
        {
                printf("Usage: ./problem2 i j k N\n");
                return 0;
        }


	int nRows = atoi(argv[1]);
	int nInnerDimension = atoi(argv[2]);
	int nCols = atoi(argv[3]);
	int num_elementsA= nRows*nInnerDimension;
	int num_elementsB=nInnerDimension*nCols;
	int num_elementsC= nRows*nCols;
	int tileSize = atoi(argv[4]);  //change this for scaling analysis
	float incTime=0; // Time for GPU
	double* hA = read_array("problem3.inp",num_elementsA);
	double* hB = read_array("problem3.inp",num_elementsB);
	double* hC = (double*) malloc(num_elementsC * sizeof(double));

	// **===-------- Modify the body of this function -----------===**
	computeOnDevice( hA, hB,hC, nRows, nInnerDimension, nCols, tileSize, &incTime);
	// **===-----------------------------------------------------------===**


	printf("%f\n%f\n%d\n%d\n%d\n",hC[num_elementsC-1],incTime,tileSize,nRows,nCols);
	// cleanup memory
	free(hA);
	free(hB);
	free(hC);

	return 0;
}


void computeOnDevice(double* hA,double* hB, double* hC, int nRows, int nInnerDimension, int nCols, int TileSize, float* incTime)
{
	hipEvent_t startEvent_inc, stopEvent_inc;
	float elapsedTime_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventRecord(startEvent_inc,0);

	double* Ad;
	hipMalloc((void**)&Ad, nRows * nInnerDimension * sizeof(double));
	hipMemcpy(Ad, hA, nRows * nInnerDimension * sizeof(double), hipMemcpyHostToDevice);

	double* Bd;
	hipMalloc((void **)&Bd, nInnerDimension * nCols * sizeof(double));
	hipMemcpy(Bd, hB, nInnerDimension * nCols * sizeof(double), hipMemcpyHostToDevice);

	double* Cd;
	hipMalloc((void **)&Cd, nRows * nCols * sizeof(double));

	dim3 dimBlock(TileSize, TileSize);

	int tempx = nRows;
	if (nInnerDimension > nRows) tempx = nInnerDimension;
	tempx = (tempx + TileSize - 1)/TileSize;
	int tempy = nCols;
	if (nInnerDimension > nCols) tempy = nInnerDimension;
	tempy = (tempy + TileSize - 1)/TileSize;

	dim3 dimGrid(tempx, tempy);

	MatMult<<<dimGrid, dimBlock, sizeof(double) * TileSize * TileSize>>>(Ad, Bd, Cd, nRows, nInnerDimension, nCols, TileSize); //, sizeof(double) * TileSize * TileSize

	hipMemcpy(hC, Cd, nRows * nCols * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	hipEventRecord(stopEvent_inc,0);
	hipEventSynchronize(stopEvent_inc);
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);
	*incTime = elapsedTime_inc;

	return;
}

