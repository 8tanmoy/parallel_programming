//Modified from:
//https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hipsolver.h>
#define MILLION 1000000L;
#include<sys/time.h>

void printMatrix(int m, int n, const float*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

int main (int argc , char* argv[])
{
    hipsolverHandle_t cusolverH;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    
    hipError_t cudaStat = hipSuccess;
    const int m = 1 << atoi(argv[1]); // number of rows and columns of A
    const int lda = m; // leading dimension of A
    float *A; // mxm matrix
    float *V; // mxm matrix of eigenvectors
    float *W; // m- vector of eigenvalues
    
    // prepare memory on the host
    A = ( float *) malloc (lda*m* sizeof ( float ));
    V = ( float *) malloc (lda*m* sizeof ( float ));
    W = ( float *) malloc (m* sizeof ( float ));
    
    // define random A
    //for(int i=0;i<lda*m;i++) A[i] = rand ()/( float ) RAND_MAX ;
    srand(time(NULL));
   
    float spf = 0.001; 
    for (int i = 0; i < m; i++)
    {
        for(int j = 0; j < m; j++)
        {
            //A[j + i*m] = spf * (((float) rand() / (RAND_MAX)) -1);
            A[j + i*m] = spf * 1.0;
        }
    }
    for (int i = 0; i < m; i++)
    {
        A[i + i*m] += (i+1)* 1.0;
    }
    for (int i = 0; i < m; i++)
    {
        for(int j = 0; j < m; j++)
        {
            A[j + i*m] = 0.5*(A[j + i*m] + A[i + j*m]);
        }
    }
	int k = 0;
    //printMatrix(m,m,A,lda,"A");	
	        
    // declare arrays on the device
    float *d_A; // mxm matrix A on the device
    float *d_W; // m- vector of eigenvalues on the device
    int * devInfo ; // info on the device
    float * d_work ; // workspace on the device
    int lwork = 0; // workspace size
    int info_gpu = 0; // info copied from device to host
    
    struct timespec start, stop;
    double accum;

    // create cusolver handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    // prepare memory on the device
    cudaStat = hipMalloc (( void **)& d_A , sizeof ( float )* lda*m);
    cudaStat = hipMalloc (( void **)& d_W , sizeof ( float )*m);
    cudaStat = hipMalloc (( void **)& devInfo , sizeof (int ));
    cudaStat = hipMemcpy (d_A ,A, sizeof ( float )* lda*m, hipMemcpyHostToDevice); // copy A->d_A
    // compute eigenvalues and eigenvectors
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    // use lower left triangle of the matrix
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    // compute buffer size and prepare workspace
    cusolver_status = hipsolverDnSsyevd_bufferSize ( cusolverH, jobz , uplo , m, d_A , lda , d_W , &lwork);
    cudaStat = hipMalloc (( void **)&d_work , sizeof ( float )* lwork);

    clock_gettime(CLOCK_REALTIME, &start);

    // compute the eigenvalues and eigenvectors for a symmetric ,
    // real mxm matrix ( only the lower left triangle af A is used )
    cusolver_status = hipsolverDnSsyevd(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork, devInfo);
    cudaStat = hipDeviceSynchronize();

    clock_gettime(CLOCK_REALTIME, &stop);
    accum = (stop.tv_sec - start.tv_sec) * 1000 + (stop.tv_nsec - start.tv_nsec)/(double)MILLION;
    //printf("Ssyevd time: %lf sec\n", accum);

    cudaStat = hipMemcpy (W, d_W , sizeof ( float )*m, hipMemcpyDeviceToHost ); // copy d_W ->W
    cudaStat = hipMemcpy (V, d_A , sizeof ( float )* lda*m, hipMemcpyDeviceToHost ); // copy d_A ->V
    cudaStat = hipMemcpy (&info_gpu , devInfo , sizeof (int), hipMemcpyDeviceToHost ); // copy devInfo -> info_gpu
    //printf (" after syevd : info_gpu = %d\n", info_gpu );
    //printf (" eigenvalues :\n"); // print first eigenvalues

    printf ("%d\t%d\t%.15lf\t%lf\n", m, k, W[k], accum);

    // free memory
    hipFree (d_A );
    hipFree (d_W );
    hipFree ( devInfo );
    hipFree ( d_work );
    hipsolverDnDestroy ( cusolverH );
    hipDeviceReset ();

    return 0;
}


 /*
 //TP: start time
     cudaEvent_t startEvent_inc, stopEvent_inc;
     float elapsedTime_inc;
     cudaEventCreate(&startEvent_inc);
     cudaEventCreate(&stopEvent_inc);
     cudaEventRecord(startEvent_inc,0);

//TP: end time
    cudaEventRecord(stopEvent_inc,0);
    cudaEventSynchronize(stopEvent_inc);
    cudaEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);
    printf("elapsed time: %f ms",elapsedTime_inc);
*/
