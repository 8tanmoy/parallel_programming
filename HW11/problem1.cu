#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

int checkResults(float*res, float* cudaRes,int length)
{
	int nDiffs=0;
	const float smallVal = .3f; // Keeping this extra high as we have repetitive addition and sequence matters
	for(int i=0; i<length; i++)
		if(fabs(cudaRes[i]-res[i])>smallVal)
			{nDiffs++;
       //printf("%f %f\n",cudaRes[i],res[i]);
      }
	return nDiffs;
}

void initializeArray(FILE* fp,float* arr, int nElements)
{
	for( int i=0; i<nElements; i++){
		int r=fscanf(fp,"%f",&arr[i]);
		if(r == EOF){
			rewind(fp);
		}
		arr[i]-=5; // This is to make the data zero mean. Otherwise we reach large numbers and lose precision
	}
}

void inclusiveScan_SEQ(float *in, float *out,int length) {
	float sum=0.f;
	for (int i =0; i < length; i++) {
		sum+=in[i];
		out[i]=sum;
	}
}

int main(int argc, char* argv[]) {
	if(argc!=2){
		printf("Usage %s N\n",argv[0]);
		return 1;
	}
	int N 			= atoi(argv[1]);
	FILE *fp 		= fopen("problem1.inp","r");
	int size 		= N * sizeof(float);
	float *in 		= (float *)malloc(size);
	float *out 		= (float *)malloc(size); 
	float *cuda_out = (float *)malloc(size);
	float time 		= 0.f;

	hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);

	initializeArray(fp,in, N);

	thrust::host_vector<float> H(N);
	for (int i = 0; i < N; i++)
	{
		H[i] = in[i];
	}
	hipEventRecord(startEvent_inc,0); // starting timing for inclusive  

	thrust::device_vector<float> D = H;
	thrust::inclusive_scan(D.begin(), D.end(), D.begin());
	thrust::copy(D.begin(), D.end(), cuda_out);

	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);

	inclusiveScan_SEQ(in, out, N);
	int nDiffs = checkResults(out, cuda_out,N);

	if(nDiffs)printf("Test Failed\n"); // This should never print
	printf("%d\n%f\n%f\n",N,cuda_out[N-1],time);
	//dynamically delete all
	return(0);
}