#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/count.h>

void computeSum( float* reference, float* idata, const unsigned int len) 
{
  reference[0] = 0;
  double total_sum = 0;
  unsigned int i;
  for( i = 0; i < len; ++i) 
  {
      total_sum += idata[i];
  }
  *reference = total_sum;
}

void initializeArray(FILE* fp,float* arr, int nElements)
{
	for( int i=0; i<nElements; i++){
		int r=fscanf(fp,"%f",&arr[i]);
		if(r == EOF){
			rewind(fp);
		}
		arr[i]-=5; // This is to make the data zero mean. Otherwise we reach large numbers and lose precision
	}
}

int main(int argc, char* argv[]) {
	if(argc!=2){
		printf("Usage %s N\n",argv[0]);
		return 1;
	}
	int N 			= atoi(argv[1]);
	FILE *fp 		= fopen("problem1.inp","r");
	int size 		= N * sizeof(float);
	float *in 		= (float *)malloc(size);
	float time 		= 0.f;

	hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);

	initializeArray(fp,in, N);

	float reference = 1.0f;  
	computeSum(&reference , in, N);

	thrust::host_vector<float> H(N);
	for (int i = 0; i < N; i++)
	{
		H[i] = in[i];
	}
	hipEventRecord(startEvent_inc,0); // starting timing for inclusive  

	thrust::device_vector<float> D = H;
	float result = thrust::reduce(D.begin(), D.end()); //, (int) 0, thrust::plus<int>());
	
	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);

	// Run accuracy test
	float epsilon = 0.3f;
	unsigned int result_regtest = (abs(result - reference) <= epsilon);

	if(!result_regtest)	printf("Test failed device: %f  host: %f\n",result,reference);

	printf("%d\n%f\n%f\n",N,result,time);
	return(0);
}